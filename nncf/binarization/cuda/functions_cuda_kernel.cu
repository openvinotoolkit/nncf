#include "hip/hip_runtime.h"
#include <ATen/ATen.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <vector>
#include <THC/THC.h>

const int CUDA_WARP_SIZE = 32;
const int CUDA_GRID_SIZE = 56; // #SM*2
const int CUDA_NUM_THREADS = 1024;

inline int GET_BLOCKS(const int N) {
    return (N + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS;
}

namespace {

__device__ bool last_block(int* counter) {
    __threadfence();

    int last = 0;
    if (threadIdx.x == 0) {
        last = atomicAdd(counter, 1);
    }

    return __syncthreads_or(last == gridDim.x - 1);
}

// support only warp size = 32
template <typename scalar_t>
__device__ void sum_warp(volatile scalar_t* sharr) {
    int tidx = threadIdx.x & 31;
    if (tidx < 16) {
        sharr[tidx] += sharr[tidx + 16];
        sharr[tidx] += sharr[tidx + 8];
        sharr[tidx] += sharr[tidx + 4];
        sharr[tidx] += sharr[tidx + 2];
        sharr[tidx] += sharr[tidx + 1];
    }
}

template <typename scalar_t>
__device__ void sumReduce(
        scalar_t* __restrict__ sh_grad,
        scalar_t sum,
        const int tidx,
        const int bidx,
        scalar_t* __restrict__ dev_tmp,
        int* __restrict__ dev_last_block_counter,
        scalar_t* __restrict__ grad) {
    sh_grad[tidx] = sum;

    __syncthreads();
    sum_warp(sh_grad + (tidx & ~(CUDA_WARP_SIZE - 1)));

    __syncthreads();
    if (tidx < CUDA_WARP_SIZE) {
        sh_grad[tidx] = tidx * CUDA_WARP_SIZE < CUDA_NUM_THREADS ? sh_grad[tidx * CUDA_WARP_SIZE] : 0;
        sum_warp(sh_grad);
        if (tidx == 0) {
            dev_tmp[bidx] = sh_grad[0];
        }
    }

    if (last_block(dev_last_block_counter)) {
        sh_grad[tidx] = tidx < gridDim.x ? dev_tmp[tidx] : 0;

        __syncthreads();
        sum_warp(sh_grad + (tidx & ~(CUDA_WARP_SIZE - 1)));

        __syncthreads();
        if (tidx < CUDA_WARP_SIZE) {
            sh_grad[tidx] = tidx * CUDA_WARP_SIZE < CUDA_NUM_THREADS ? sh_grad[tidx * CUDA_WARP_SIZE] : 0;
            sum_warp(sh_grad);
            if (tidx == 0) {
                grad[0] = sh_grad[0];
            }
        }
    }
}


template <typename scalar_t>
__global__ void wb_cuda_scale_calc_kernel(
        const scalar_t* __restrict__ input,
        scalar_t* __restrict__ scale_output,
        scalar_t* __restrict__ dev_tmp,
        int* __restrict__ dev_last_block_counter,
        const int64_t total_elements_count) {
    const int tidx = threadIdx.x;
    const int bidx = blockIdx.x;
    const int gtidx = bidx * CUDA_NUM_THREADS + tidx;
    const int grid_size = CUDA_NUM_THREADS * gridDim.x;

    scalar_t sum = 0;
    for (int i = gtidx; i < total_elements_count; i += grid_size) {
        sum += abs(*(input + i));
    }

    sum /= total_elements_count;

    __shared__ scalar_t sh_mem[CUDA_NUM_THREADS];
    sumReduce<scalar_t>(sh_mem, sum, tidx, bidx, dev_tmp, dev_last_block_counter, scale_output);
}

template <typename scalar_t>
__global__ void wb_cuda_binarize_kernel(
        scalar_t* __restrict__ output,
        const scalar_t* __restrict__ input,
        const scalar_t* __restrict__ scale,
        const int64_t scale_count,
        const int64_t elements_per_scale,
        const int64_t size) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < size) {
        int64_t scale_idx = static_cast<int64_t>(idx / elements_per_scale) % scale_count;
        scalar_t scale_element = *(scale + scale_idx);
        *(output + idx) = (*(input + idx) > 0) ? scale_element : -scale_element;
    }
}


template <typename scalar_t>
__global__ void ab_cuda_forward_kernel(
        scalar_t* __restrict__ output,
        const scalar_t* __restrict__ input,
        const scalar_t* __restrict__ scale,
        const scalar_t* __restrict__ thresholds,
        const int64_t threshold_count,
        const int64_t contiguous_elements_per_threshold,
        const int64_t size) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < size) {
        int64_t threshold_idx = static_cast<int64_t>(idx / contiguous_elements_per_threshold) % threshold_count;
        scalar_t threshold_element = (*(thresholds + threshold_idx)) * (*scale);
        *(output + idx) = (*(input + idx) > threshold_element) ? (*scale) : 0;
    }
}


template <typename scalar_t>
__global__ void ab_cuda_grad_input_kernel(
        scalar_t* __restrict__ grad_input,
        const scalar_t* __restrict__ grad_output,
        const scalar_t* __restrict__ input,
        const scalar_t* __restrict__ scale,
        const int64_t size) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < size) {
        const scalar_t input_element = *(input + idx);
        *(grad_input + idx) = (input_element > 0 && input_element < *scale) ? *(grad_output + idx) : 0;
    }
}

template <typename scalar_t>
__global__ void ab_cuda_grad_scale_kernel(
        scalar_t* __restrict__ grad_scale,
        const scalar_t* __restrict__ grad_output,
        const scalar_t* __restrict__ output,
        const scalar_t* __restrict__ input,
        const scalar_t* __restrict__ scale,
        scalar_t* __restrict__ dev_tmp,
        int* __restrict__ dev_last_block_counter,
        const int64_t total_elements_count) {
    const int tidx = threadIdx.x;
    const int bidx = blockIdx.x;
    const int gtidx = bidx * CUDA_NUM_THREADS + tidx;
    const int grid_size = CUDA_NUM_THREADS * gridDim.x;

    scalar_t sum = 0;
    for (int i = gtidx; i < total_elements_count; i += grid_size) {
        scalar_t err_element = (*(output + i) - *(input + i)) / *scale;
        scalar_t grad_element = *(grad_output + i);
        sum += (*(input + i) < *scale) ? err_element * grad_element : grad_element;
    }

    __shared__ scalar_t sh_mem[CUDA_NUM_THREADS];
    sumReduce<scalar_t>(sh_mem, sum, tidx, bidx, dev_tmp, dev_last_block_counter, grad_scale);
}

template <typename scalar_t>
__global__ void ab_cuda_grad_thresholds_kernel(
        scalar_t* __restrict__ grad_thresholds,
        const scalar_t* __restrict__ grad_output,
        const scalar_t* __restrict__ input,
        const scalar_t* __restrict__ scale,
        scalar_t* __restrict__ dev_tmp,
        int* __restrict__ dev_last_block_counter,
        int64_t total_elements_per_threshold,
        int64_t contiguous_elements_per_threshold,
        int64_t threshold_count,
        int64_t channel_offset) {
    const int tidx = threadIdx.x;
    const int bidx = blockIdx.x;
    const int gtidx = bidx * CUDA_NUM_THREADS + tidx;
    const int grid_size = CUDA_NUM_THREADS * gridDim.x;

    scalar_t sum = 0;
    for (int i = gtidx; i < total_elements_per_threshold; i += grid_size) {
        // i is the global thread index - need to calculate the input array index
        // that belongs to a specific scale index from i. Will do this by treating i
        // as the index in a non-existing array where input values belonging to a single
        // scale have a contiguous block layout, but will recalculate actual index into the
        // input/output array based on the fact that the values belonging to a single scale
        // in reality have interleaved block layout, with a spacing between the blocks
        // equal to channel_offset
        int actual_idx = (i / contiguous_elements_per_threshold) * channel_offset + (i % contiguous_elements_per_threshold);
        scalar_t input_element = *(input + actual_idx);
        if (input_element < *scale && input_element > 0)
        {
            sum += -*(grad_output + actual_idx);
        }
    }

    __shared__ scalar_t sh_mem[CUDA_NUM_THREADS];
    sumReduce<scalar_t>(sh_mem, sum, tidx, bidx, dev_tmp, dev_last_block_counter, grad_thresholds);
}

}

at::Tensor wb_cuda_forward(
        at::Tensor input,
        bool per_channel) {
    const auto quantized_elements_count = input.numel();

    int64_t elements_per_scale = 0;
    int64_t scale_count = per_channel ? input.size(0) : 1;
    int64_t input_elements_count = input.numel();

    auto scale = at::zeros({scale_count}, input.options());
    elements_per_scale = input_elements_count / input.size(0);

    auto grid_size = std::min(GET_BLOCKS(elements_per_scale), CUDA_GRID_SIZE);
    auto dev_tmp = at::empty({grid_size}, input.options());
    auto dev_last_block_counter = at::zeros({1},  at::device(input.options().device()).dtype(at::kInt));


    auto output = at::empty_like(input);

    for (int ch_idx = 0; ch_idx < scale_count; ch_idx++)
    {
        AT_DISPATCH_FLOATING_TYPES(input.type(), "wb_cuda_forward_scale", ([&] {
          wb_cuda_scale_calc_kernel<scalar_t><<<grid_size, CUDA_NUM_THREADS, 0, at::cuda::getCurrentCUDAStream()>>>(
              input.data<scalar_t>() + ch_idx * elements_per_scale,
              scale.data<scalar_t>() + ch_idx,
              dev_tmp.data<scalar_t>(),
              dev_last_block_counter.data<int>(),
              elements_per_scale);
        }));
        dev_tmp.fill_(0.0);
        dev_last_block_counter.fill_(0);
    }

    AT_DISPATCH_FLOATING_TYPES(input.type(), "wb_cuda_forward_binarize", ([&] {
      wb_cuda_binarize_kernel<scalar_t><<<GET_BLOCKS(input_elements_count), CUDA_NUM_THREADS, 0, at::cuda::getCurrentCUDAStream()>>>(
          output.data<scalar_t>(),
          input.data<scalar_t>(),
          scale.data<scalar_t>(),
          scale_count,
          elements_per_scale,
          input_elements_count
          );
    }));

    return output;
}


at::Tensor ab_cuda_forward(
        at::Tensor input,
        at::Tensor scale,
        at::Tensor thresholds) {
    const auto quantized_elements_count = input.numel();

    int64_t input_elements_count = input.numel();
    int64_t threshold_count = thresholds.numel();
    TORCH_CHECK(input.size(1) == threshold_count, "Threshold count is not equal to activations channel count");
    int64_t contiguous_elements_per_threshold = input_elements_count / input.size(0) / input.size(1);

    auto output = at::empty_like(input);

    AT_DISPATCH_FLOATING_TYPES(input.type(), "ab_cuda_forward", ([&] {
      ab_cuda_forward_kernel<scalar_t><<<GET_BLOCKS(input_elements_count), CUDA_NUM_THREADS, 0, at::cuda::getCurrentCUDAStream()>>>(
          output.data<scalar_t>(),
          input.data<scalar_t>(),
          scale.data<scalar_t>(),
          thresholds.data<scalar_t>(),
          threshold_count,
          contiguous_elements_per_threshold,
          input_elements_count
          );
    }));

    return output;
}


std::vector<at::Tensor> ab_cuda_backward(
        at::Tensor grad_output,
        at::Tensor input,
        at::Tensor scale,
        at::Tensor output) {
    int64_t input_elements_count = input.numel();
    int64_t threshold_count = input.size(1);
    int64_t channel_offset = input.numel() / input.size(0);

    std::vector<int64_t> threshold_shape(input.dim());
    for (int64_t dim_idx = 0; dim_idx < input.dim(); dim_idx++)
    {
        if (dim_idx != 1)
        {
            threshold_shape[dim_idx] = 1;
        }
        else
        {
            threshold_shape[dim_idx] = input.size(dim_idx);
        }
    }

    auto grad_input = at::empty_like(input);
    auto grad_scale = at::empty_like(scale);
    auto grad_thresholds = at::empty(threshold_shape, input.options());

    int64_t total_elements_per_threshold = input.numel() / threshold_count;
    int64_t contiguous_elements_per_threshold = input_elements_count / input.size(0) / input.size(1);

    AT_DISPATCH_FLOATING_TYPES(input.type(), "ab_cuda_backward", ([&] {
      ab_cuda_grad_input_kernel<scalar_t><<<GET_BLOCKS(input_elements_count), CUDA_NUM_THREADS, 0, at::cuda::getCurrentCUDAStream()>>>(
          grad_input.data<scalar_t>(),
          grad_output.data<scalar_t>(),
          input.data<scalar_t>(),
          scale.data<scalar_t>(),
          input_elements_count
          );
    }));

    auto grid_size = std::min(GET_BLOCKS(input_elements_count), CUDA_GRID_SIZE);
    auto dev_tmp = at::empty({grid_size}, grad_output.options());
    auto dev_last_block_counter = at::zeros({1},  at::device(grad_output.options().device()).dtype(at::kInt));


    AT_DISPATCH_FLOATING_TYPES(input.type(), "ab_cuda_backward", ([&] {
          ab_cuda_grad_scale_kernel<scalar_t><<<grid_size, CUDA_NUM_THREADS, 0, at::cuda::getCurrentCUDAStream()>>>(
              grad_scale.data<scalar_t>(),
              grad_output.data<scalar_t>(),
              output.data<scalar_t>(),
              input.data<scalar_t>(),
              scale.data<scalar_t>(),
              dev_tmp.data<scalar_t>(),
              dev_last_block_counter.data<int>(),
              input_elements_count);
        }));

    grid_size = std::min(GET_BLOCKS(total_elements_per_threshold), CUDA_GRID_SIZE);
    dev_tmp = at::empty({grid_size}, grad_output.options());
    dev_last_block_counter = at::zeros({1},  at::device(grad_output.options().device()).dtype(at::kInt));

    // Same concept as for per activation channel quantization
    for (int64_t ch_idx = 0; ch_idx < threshold_count; ch_idx++)
    {
        auto init_element_offset = contiguous_elements_per_threshold * ch_idx;
        AT_DISPATCH_FLOATING_TYPES(input.type(), "ab_cuda_backward", ([&] {
          ab_cuda_grad_thresholds_kernel<scalar_t><<<grid_size, CUDA_NUM_THREADS, 0, at::cuda::getCurrentCUDAStream()>>>(
              grad_thresholds.data<scalar_t>() + ch_idx,
              grad_output.data<scalar_t>() + init_element_offset,
              input.data<scalar_t>() + init_element_offset,
              scale.data<scalar_t>(),
              dev_tmp.data<scalar_t>(),
              dev_last_block_counter.data<int>(),
              total_elements_per_threshold,
              contiguous_elements_per_threshold,
              threshold_count,
              channel_offset);
        }));
        dev_tmp.fill_(0.0);
        dev_last_block_counter.fill_(0);
    }
    return {grad_input, grad_scale, grad_thresholds};
}

