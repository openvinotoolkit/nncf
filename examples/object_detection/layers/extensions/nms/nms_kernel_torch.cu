#include "hip/hip_runtime.h"
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>

#include "cuda_helpers.h"
#include "nms.h"

#include <ATen/ATen.h>
// #include "macros.h"

namespace vision {
namespace ops {

namespace {

int const threadsPerBlock = sizeof(unsigned long long) * 8;

template <typename T>
__device__ inline bool devIoU(
    T const* const a,
    T const* const b,
    const float threshold) {
  T left = max(a[0], b[0]), right = min(a[2], b[2]);
  T top = max(a[1], b[1]), bottom = min(a[3], b[3]);
  T width = max(right - left, (T)0), height = max(bottom - top, (T)0);
  T interS = width * height;
  T Sa = (a[2] - a[0]) * (a[3] - a[1]);
  T Sb = (b[2] - b[0]) * (b[3] - b[1]);
  return (interS / (Sa + Sb - interS)) > threshold;
}

template <typename T>
__global__ void nms_kernel_impl(
    int n_boxes,
    double iou_threshold,
    const T* dev_boxes,
    unsigned long long* dev_mask) {
  const int row_start = blockIdx.y;
  const int col_start = blockIdx.x;

  if (row_start > col_start)
    return;

  const int row_size =
      min(n_boxes - row_start * threadsPerBlock, threadsPerBlock);
  const int col_size =
      min(n_boxes - col_start * threadsPerBlock, threadsPerBlock);

  __shared__ T block_boxes[threadsPerBlock * 4];
  if (threadIdx.x < col_size) {
    block_boxes[threadIdx.x * 4 + 0] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 4 + 0];
    block_boxes[threadIdx.x * 4 + 1] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 4 + 1];
    block_boxes[threadIdx.x * 4 + 2] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 4 + 2];
    block_boxes[threadIdx.x * 4 + 3] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 4 + 3];
  }
  __syncthreads();

  if (threadIdx.x < row_size) {
    const int cur_box_idx = threadsPerBlock * row_start + threadIdx.x;
    const T* cur_box = dev_boxes + cur_box_idx * 4;
    int i = 0;
    unsigned long long t = 0;
    int start = 0;
    if (row_start == col_start) {
      start = threadIdx.x + 1;
    }
    for (i = start; i < col_size; i++) {
      if (devIoU<T>(cur_box, block_boxes + i * 4, iou_threshold)) {
        t |= 1ULL << i;
      }
    }
    const int col_blocks = ceil_div(n_boxes, threadsPerBlock);
    dev_mask[cur_box_idx * col_blocks + col_start] = t;
  }
}

} // namespace

at::Tensor nms_gpu(
    const at::Tensor& dets,
    const at::Tensor& scores,
    double iou_threshold, int64_t top_k) {
  TORCH_CHECK(dets.is_cuda(), "dets must be a CUDA tensor");
  TORCH_CHECK(scores.is_cuda(), "scores must be a CUDA tensor");

  TORCH_CHECK(
      dets.dim() == 2, "boxes should be a 2d tensor, got ", dets.dim(), "D");
  TORCH_CHECK(
      dets.size(1) == 4,
      "boxes should have 4 elements in dimension 1, got ",
      dets.size(1));
  TORCH_CHECK(
      scores.dim() == 1,
      "scores should be a 1d tensor, got ",
      scores.dim(),
      "D");
  TORCH_CHECK(
      dets.size(0) == scores.size(0),
      "boxes and scores should have same number of elements in ",
      "dimension 0, got ",
      dets.size(0),
      " and ",
      scores.size(0))

#if defined(WITH_CUDA) || defined(WITH_HIP)
  at::cuda::CUDAGuard device_guard(dets.device());
#else
  TORCH_CHECK(false, "Not compiled with GPU support");
#endif

  if (dets.numel() == 0) {
    return at::empty({0}, dets.options().dtype(at::kLong));
  }

  auto order_t = std::get<1>(scores.sort(0, /* descending=*/true));
  auto dets_sorted = dets.index_select(0, order_t).contiguous();

  int dets_num = dets.size(0);

  const int col_blocks = ceil_div(dets_num, threadsPerBlock);

  at::Tensor mask =
      at::empty({dets_num * col_blocks}, dets.options().dtype(at::kLong));

  dim3 blocks(col_blocks, col_blocks);
  dim3 threads(threadsPerBlock);
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      dets_sorted.scalar_type(), "nms_cuda", [&] {
        nms_kernel_impl<scalar_t><<<blocks, threads, 0, stream>>>(
            dets_num,
            iou_threshold,
            dets_sorted.data_ptr<scalar_t>(),
            (unsigned long long*)mask.data_ptr<int64_t>());
      });

  at::Tensor mask_cpu = mask.to(at::kCPU);
  unsigned long long* mask_host =
      (unsigned long long*)mask_cpu.data_ptr<int64_t>();

  std::vector<unsigned long long> remv(col_blocks);
  memset(&remv[0], 0, sizeof(unsigned long long) * col_blocks);

  at::Tensor keep =
      at::empty({dets_num}, dets.options().dtype(at::kLong).device(at::kCPU));
  int64_t* keep_out = keep.data_ptr<int64_t>();

  int num_to_keep = 0;
  for (int i = 0; i < dets_num; i++) {
    int nblock = i / threadsPerBlock;
    int inblock = i % threadsPerBlock;

    if (!(remv[nblock] & (1ULL << inblock))) {
      keep_out[num_to_keep++] = i;
      unsigned long long* p = mask_host + i * col_blocks;
      for (int j = nblock; j < col_blocks; j++) {
        remv[j] |= p[j];
      }
    }
  }

  AT_CUDA_CHECK(hipGetLastError());
  return order_t.index(
      {keep.narrow(/*dim=*/0, /*start=*/0, /*length=*/num_to_keep)
           .to(order_t.device(), keep.scalar_type())});
}

} // namespace ops
} // namespace vision
